
#include "cuda_type.h"
#include "ntsCUDA.hpp"

#if CUDA_ENABLE
#include "ntsCUDADistKernel.cuh"
#include "ntsCUDAFuseKernel.cuh"
#include "ntsCUDATransferKernel.cuh"

#endif

#if CUDA_ENABLE
#define CHECK_CUDA_RESULT(N)                                                \
  {                                                                         \
    hipError_t result = N;                                                 \
    if (result != 0) {                                                      \
      printf("CUDA call on line %d returned error %d\n", __LINE__, result); \
      exit(1);                                                              \
    }                                                                       \
  }
#endif

void* getDevicePointer(void* host_data_to_device) {
#if CUDA_ENABLE
  void* dev_host_data_to_device;
  CHECK_CUDA_RESULT(hipHostGetDevicePointer(&dev_host_data_to_device, host_data_to_device, 0));
  return dev_host_data_to_device;
#else
  printf("CUDA DISABLED getDevicePointer\n");
  exit(0);
#endif
}

void* cudaMallocPinned(long size_of_bytes) {
#if CUDA_ENABLE
  void* data = NULL;
  CHECK_CUDA_RESULT(hipHostAlloc(&data, size_of_bytes, hipHostMallocMapped));
  return data;
#else
  printf("CUDA DISABLED cudaMallocPinned\n");
  exit(0);
#endif
}

void* cudaMallocGPU(long size_of_bytes) {
#if CUDA_ENABLE
  void* data = NULL;
  CHECK_CUDA_RESULT(hipMalloc(&data, size_of_bytes));
  //       printf("malloc finished\n");
  return data;
#else
  printf("CUDA DISABLED cudaMallocGPU\n");
  exit(0);
#endif
}

Cuda_Stream::Cuda_Stream() {
#if CUDA_ENABLE
  CHECK_CUDA_RESULT(hipStreamCreate(&stream));
#else
  printf("CUDA DISABLED Cuda_Stream::Cuda_Stream\n");
  exit(0);
#endif
}

Cuda_Stream::~Cuda_Stream() {
#if CUDA_ENABLE
  printf("ntsCUDAGRAPHOp.cu line 68: call hip_stream ~Destory_Stream()\n");
  size_t gpu_total_size;
  size_t gpu_free_size;
  hipError_t cuda_status = hipMemGetInfo(&gpu_free_size, &gpu_total_size);
  printf("before destroy stream GPU mem: %.3f-%.3f\n", gpu_total_size * 1.0 / 1024 / 1024,
         gpu_free_size * 1.0 / 1024 / 1024);
  CHECK_CUDA_RESULT(hipStreamDestroy(stream));
  cuda_status = hipMemGetInfo(&gpu_free_size, &gpu_total_size);
  printf("after destroy stream GPU mem: %.3f-%.3f\n", gpu_total_size * 1.0 / 1024 / 1024,
         gpu_free_size * 1.0 / 1024 / 1024);
  printf("ntsCUDAGRAPHOp.cu line 70: call hip_stream is done\n");
#else
  printf("CUDA DISABLED Cuda_Stream::~Cuda_Stream\n");
  exit(0);
#endif
}

void Cuda_Stream::destory_Stream() {
#if CUDA_ENABLE
  CHECK_CUDA_RESULT(hipStreamDestroy(stream));
  printf("destory_Stream is done\n");
#else
  printf("CUDA DISABLED Cuda_Stream::Cuda_Stream\n");
  exit(0);
#endif
}
inline hipStream_t Cuda_Stream::getStream() {
#if CUDA_ENABLE
  return stream;
#else
  printf("CUDA DISABLED Cuda_Stream::getStream\n");
  exit(0);
#endif
}

void ResetDevice() {
#if CUDA_ENABLE
  hipDeviceReset();
#else
  printf("CUDA DISABLED ResetDevice\n");
  exit(0);
#endif
}
void Cuda_Stream::CUDA_DEVICE_SYNCHRONIZE() {
#if CUDA_ENABLE
  hipStreamSynchronize(stream);
#else
  printf("CUDA DISABLED Cuda_Stream::CUDA_DEVICE_SYNCHRONIZE\n");
  exit(0);
#endif
}

void Cuda_Stream::move_result_out(float* output, float* input, VertexId_CUDA src, VertexId_CUDA dst, int feature_size,
                                  bool sync) {
#if CUDA_ENABLE
  CHECK_CUDA_RESULT(hipMemcpyAsync(output, input, ((long)(dst - src)) * feature_size * (sizeof(int)),
                                    hipMemcpyDeviceToHost, stream));
#else
  printf("CUDA DISABLED Cuda_Stream::move_result_out\n");
  exit(0);
#endif
}
void Cuda_Stream::move_data_in(float* d_pointer, float* h_pointer, VertexId_CUDA start, VertexId_CUDA end,
                               int feature_size, bool sync) {
#if CUDA_ENABLE
  CHECK_CUDA_RESULT(hipMemcpyAsync(d_pointer, h_pointer, ((long)(end - start)) * feature_size * (sizeof(float)),
                                    hipMemcpyHostToDevice, stream));
#else
  printf("CUDA DISABLED Cuda_Stream::move_data_in\n");
  exit(0);
#endif
}
void Cuda_Stream::move_edge_in(VertexId_CUDA* d_pointer, VertexId_CUDA* h_pointer, VertexId_CUDA start,
                               VertexId_CUDA end, int feature_size, bool sync) {
#if CUDA_ENABLE
  CHECK_CUDA_RESULT(hipMemcpyAsync(d_pointer, h_pointer,
                                    ((long)(end - start)) * feature_size * (sizeof(VertexId_CUDA)),
                                    hipMemcpyHostToDevice, stream));
#else
  printf("CUDA DISABLED Cuda_Stream::move_edge_in\n");
  exit(0);
#endif
}
void Cuda_Stream::aggregate_comm_result(float* aggregate_buffer, float* input_buffer, VertexId_CUDA data_size,
                                        int feature_size, int partition_offset, bool sync) {
#if CUDA_ENABLE
  aggregate_data_buffer<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(aggregate_buffer, input_buffer, data_size,
                                                                          feature_size, partition_offset, sync);
#else
  printf("CUDA DISABLED Cuda_Stream::aggregate_comm_result\n");
  exit(0);
#endif
}

void Cuda_Stream::aggregate_comm_result_debug(float* aggregate_buffer, float* input_buffer, VertexId_CUDA data_size,
                                              VertexId_CUDA feature_size, VertexId_CUDA partition_start,
                                              VertexId_CUDA partition_end, bool sync) {
#if CUDA_ENABLE
  aggregate_data_buffer_debug<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
      aggregate_buffer, input_buffer, data_size, feature_size, partition_start, partition_end, sync);
#else
  printf("CUDA DISABLED Cuda_Stream::aggregate_comm_result_debug\n");
  exit(0);
#endif
}

void Cuda_Stream::deSerializeToGPU(float* input_gpu_buffer, float* input_buffer, VertexId_CUDA data_size,
                                   VertexId_CUDA feature_size, VertexId_CUDA partition_start,
                                   VertexId_CUDA partition_end, bool sync) {
#if CUDA_ENABLE
  deSerializeToGPUkernel<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
      input_gpu_buffer, input_buffer, data_size, feature_size, partition_start, partition_end, sync);
#else
  printf("CUDA DISABLED Cuda_Stream::deSerializeToGPU\n");
  exit(0);
#endif
}
void Cuda_Stream::Gather_By_Dst_From_Src(float* input, float* output, float* weight_forward,        // data
                                         VertexId_CUDA* row_indices, VertexId_CUDA* column_offset,  // graph
                                         VertexId_CUDA src_start, VertexId_CUDA src_end, VertexId_CUDA dst_start,
                                         VertexId_CUDA dst_end, VertexId_CUDA edges, VertexId_CUDA batch_size,
                                         VertexId_CUDA feature_size, bool with_weight, bool tensor_weight) {
#if CUDA_ENABLE
  if (with_weight) {
    if (tensor_weight) {
      //		aggregate_kernel_from_src_tensor_weight<float,VertexId_CUDA><<<BLOCK_SIZE,THREAD_SIZE,0,stream>>>(
      //			row_indices, column_offset, input, output, weight_forward,
      //				src_start, dst_start, batch_size, feature_size);
      printf("aggregate_kernel_from_src_tensor_weight_optim_nts");
    } else {
      aggregate_kernel_from_src_with_weight<float, VertexId_CUDA><<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
          row_indices, column_offset, input, output, weight_forward, src_start, dst_start, batch_size, feature_size);
    }
  } else {
    aggregate_kernel_from_src_without_weight<float, VertexId_CUDA><<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
        row_indices, column_offset, input, output, weight_forward, src_start, dst_start, batch_size, feature_size);
  }
#else
  printf("CUDA DISABLED Cuda_Stream::Gather_By_Dst_From_Src\n");
  exit(0);
#endif
}
void Cuda_Stream::Gather_By_Dst_From_Src_Optim(float* input, float* output, float* weight_forward,  // data
                                               VertexId_CUDA* row_indices, VertexId_CUDA* column_offset,
                                               VertexId_CUDA src_start, VertexId_CUDA src_end, VertexId_CUDA dst_start,
                                               VertexId_CUDA dst_end, VertexId_CUDA edges, VertexId_CUDA batch_size,
                                               VertexId_CUDA feature_size, bool with_weight, bool tensor_weight) {
#if CUDA_ENABLE
  if (with_weight) {
    if (tensor_weight) {
      //		aggregate_kernel_from_src_tensor_weight_optim_nts<<<CUDA_NUM_BLOCKS,CUDA_NUM_THREADS,0,stream>>>(
      //			row_indices, column_offset, input, output, weight_forward,
      //				src_start,src_end, dst_start,dst_end,edges, batch_size, feature_size);
      printf("aggregate_kernel_from_src_tensor_weight_optim_nts is a legacy implementation\n");
      exit(0);
    } else {
      aggregate_kernel_from_src_with_weight_optim_nts<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
          row_indices, column_offset, input, output, weight_forward, src_start, src_end, dst_start, dst_end, edges,
          batch_size, feature_size);
    }
  } else {
    aggregate_kernel_from_src_without_weight_optim_nts<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
        row_indices, column_offset, input, output, weight_forward, src_start, src_end, dst_start, dst_end, edges,
        batch_size, feature_size);
  }
#else
  printf("CUDA DISABLED Cuda_Stream::Gather_By_Dst_From_Src_Optim\n");
  exit(0);
#endif
}

void Cuda_Stream::Gather_By_Src_From_Dst_Optim(float* input, float* output, float* weight_forward,  // data
                                               VertexId_CUDA* row_offset, VertexId_CUDA* column_indices,
                                               VertexId_CUDA src_start, VertexId_CUDA src_end, VertexId_CUDA dst_start,
                                               VertexId_CUDA dst_end, VertexId_CUDA edges, VertexId_CUDA batch_size,
                                               VertexId_CUDA feature_size, bool with_weight, bool tensor_weight) {
#if CUDA_ENABLE
  if (with_weight) {
    if (tensor_weight) {
      //		aggregate_kernel_from_dst_tensor_weight_optim_nts<<<CUDA_NUM_BLOCKS,CUDA_NUM_THREADS,0,stream>>>(
      //			row_offset, column_indices, input, output, weight_forward,
      //				src_start,src_end, dst_start,dst_end,edges, batch_size, feature_size);
      printf("aggregate_kernel_from_dst_tensor_weight_optim_nts is a legacy implementation\n");
      exit(0);
    } else {
      aggregate_kernel_from_dst_with_weight_optim_nts<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
          row_offset, column_indices, input, output, weight_forward, src_start, src_end, dst_start, dst_end, edges,
          batch_size, feature_size);
    }
  } else {
    aggregate_kernel_from_dst_without_weight_optim_nts<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
        row_offset, column_indices, input, output, weight_forward, src_start, src_end, dst_start, dst_end, edges,
        batch_size, feature_size);
  }
#else
  printf("CUDA DISABLED Gather_By_Src_From_Dst_Optim\n");
  exit(0);
#endif
}

void Cuda_Stream::Gather_By_Src_From_Dst(float* input, float* output, float* weight_forward,        // data
                                         VertexId_CUDA* row_offset, VertexId_CUDA* column_indices,  // graph
                                         VertexId_CUDA src_start, VertexId_CUDA src_end, VertexId_CUDA dst_start,
                                         VertexId_CUDA dst_end, VertexId_CUDA edges, VertexId_CUDA batch_size,
                                         VertexId_CUDA feature_size, bool with_weight, bool tensor_weight) {
#if CUDA_ENABLE
  // printf("CUDA_DEBUGE_INFO:FORWARD RUN_SYNC with \t BLOCK_SIZE:%d\tTHREAD_SIZE:%d\n",BLOCK_SIZE,THREAD_SIZE);
  if (with_weight) {
    if (tensor_weight) {
      printf("aggregate_kernel_from_dst_tensor_weight is a legacy implementation\n");
      exit(0);
    } else {
      aggregate_kernel_from_dst_with_weight<float, VertexId_CUDA><<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
          row_offset, column_indices, input, output, weight_forward, src_start, dst_start, batch_size, feature_size);
    }
  } else {
    aggregate_kernel_from_dst_without_weight<float, VertexId_CUDA><<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
        row_offset, column_indices, input, output, weight_forward, src_start, dst_start, batch_size, feature_size);
  }
#else
  printf("CUDA DISABLED Cuda_Stream::Gather_By_Src_From_Dst\n");
  exit(0);
#endif
}

void Cuda_Stream::Scatter_Grad_Back_To_Message(float* input, float* message_grad,  // data
                                               VertexId_CUDA* row_indices, VertexId_CUDA* column_offset,
                                               VertexId_CUDA src_start, VertexId_CUDA src_end, VertexId_CUDA dst_start,
                                               VertexId_CUDA dst_end, VertexId_CUDA edges, VertexId_CUDA batch_size,
                                               VertexId_CUDA feature_size, bool with_weight) {
#if CUDA_ENABLE
  // printf("CUDA_DEBUGE_INFO:FORWARD RUN_SYNC with \t BLOCK_SIZE:%d\tTHREAD_SIZE:%d\n",BLOCK_SIZE,THREAD_SIZE);
  if (with_weight) {
    printf("tensor_weight Scatter_Grad_Back_To_Weight not implemented\n");
    exit(0);
  } else {
    scatter_grad_back_to_messaage<float, VertexId_CUDA><<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
        row_indices, column_offset, input, message_grad, src_start, dst_start, batch_size, feature_size);
  }
#else
  printf("CUDA DISABLED Cuda_Stream::Scatter_Grad_Back_To_Message\n");
  exit(0);
#endif
}

void Cuda_Stream::Scatter_Src_Mirror_to_Msg(float* message, float* src_mirror_feature,  // data
                                            VertexId_CUDA* row_indices, VertexId_CUDA* column_offset,
                                            VertexId_CUDA* mirror_index, VertexId_CUDA batch_size,
                                            VertexId_CUDA feature_size) {
#if CUDA_ENABLE
  scatter_src_mirror_to_msg<float, VertexId_CUDA><<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
      message, src_mirror_feature, row_indices, column_offset, mirror_index, batch_size, feature_size);
#else
  printf("CUDA DISABLED Cuda_Stream::Scatter_Src_Mirror_to_Msg\n");
  exit(0);
#endif
}

void Cuda_Stream::Gather_Msg_To_Src_Mirror(float* src_mirror_feature, float* message,  // data
                                           VertexId_CUDA* row_indices, VertexId_CUDA* column_offset,
                                           VertexId_CUDA* mirror_index, VertexId_CUDA batch_size,
                                           VertexId_CUDA feature_size) {
#if CUDA_ENABLE
  // printf("CUDA_DEBUGE_INFO:FORWARD RUN_SYNC with \t BLOCK_SIZE:%d\tfeature_size:%d\n",BLOCK_SIZE,feature_size);
  gather_msg_to_src_mirror<float, VertexId_CUDA><<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
      src_mirror_feature, message, row_indices, column_offset, mirror_index, batch_size, feature_size);
#else
  printf("CUDA DISABLED Cuda_Stream::Gather_Msg_To_Src_Mirror\n");
  exit(0);
#endif
}

void Cuda_Stream::Scatter_Dst_to_Msg(float* message, float* dst_feature,  // data
                                     VertexId_CUDA* row_indices, VertexId_CUDA* column_offset, VertexId_CUDA batch_size,
                                     VertexId_CUDA feature_size) {
#if CUDA_ENABLE
  // printf("CUDA_DEBUGE_INFO:FORWARD RUN_SYNC with \t BLOCK_SIZE:%d\tfeature_size:%d\n",BLOCK_SIZE,feature_size);
  scatter_dst_to_msg<float, VertexId_CUDA><<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
      message, dst_feature, row_indices, column_offset, batch_size, feature_size);
#else
  printf("CUDA DISABLED Cuda_Stream::Scatter_Dst_to_Msg\n");
  exit(0);
#endif
}

void Cuda_Stream::Gather_Msg_to_Dst(float* dst_feature, float* message,  // data
                                    VertexId_CUDA* row_indices, VertexId_CUDA* column_offset, VertexId_CUDA batch_size,
                                    VertexId_CUDA feature_size) {
#if CUDA_ENABLE
  // printf("CUDA_DEBUGE_INFO:FORWARD RUN_SYNC with \t BLOCK_SIZE:%d\tfeature_size:%d\n",BLOCK_SIZE,feature_size);
  gather_msg_to_dst<float, VertexId_CUDA><<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
      dst_feature, message, row_indices, column_offset, batch_size, feature_size);
#else
  printf("CUDA DISABLED Cuda_Stream::Gather_Msg_to_Dst\n");
  exit(0);
#endif
}

void Cuda_Stream::zero_copy_feature_move_gpu(float* dev_feature, float* host_pinned_feature, VertexId_CUDA* src_vertex,
                                             VertexId_CUDA feature_size, VertexId_CUDA vertex_size) {
#if CUDA_ENABLE
  zero_copy_feature_move_gpu_kernel<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
      dev_feature, host_pinned_feature, src_vertex, feature_size, vertex_size);
  this->CUDA_DEVICE_SYNCHRONIZE();
#else
  printf("CUDA DISABLED Cuda_Stream::zero_copy_feature_move_gpu\n");
  exit(0);
#endif
}

void Cuda_Stream::zero_copy_feature_move_gpu_cache(float* dev_feature, float* host_pinned_feature,
                                                   VertexId_CUDA* src_vertex, VertexId_CUDA feature_size,
                                                   VertexId_CUDA vertex_size, VertexId_CUDA* local_idx) {
#if CUDA_ENABLE
  zero_copy_feature_move_gpu_cache_kernel<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
      dev_feature, host_pinned_feature, src_vertex, feature_size, vertex_size, local_idx);
  this->CUDA_DEVICE_SYNCHRONIZE();
#else
  printf("CUDA DISABLED Cuda_Stream::zero_copy_feature_move_gpu\n");
  exit(0);
#endif
}

void Cuda_Stream::gather_feature_from_gpu_cache(float* dev_feature, float* dev_cache_feature, VertexId_CUDA* src_vertex,
                                                VertexId_CUDA feature_size, VertexId_CUDA vertex_size,
                                                VertexId_CUDA* local_idx, VertexId_CUDA* cache_node_hashmap) {
  //  std::vector<int>& local_idx, std::vector<int>& cache_node_hashmap) {
#if CUDA_ENABLE
  gather_feature_from_gpu_cache_kernel<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
      dev_feature, dev_cache_feature, src_vertex, feature_size, vertex_size, local_idx, cache_node_hashmap);
  this->CUDA_DEVICE_SYNCHRONIZE();
#else
  printf("CUDA DISABLED Cuda_Stream::zero_copy_feature_move_gpu\n");
  exit(0);
#endif
}

void Cuda_Stream::global_copy_mulilabel_move_gpu(long* dev_label, long* global_dev_label, VertexId_CUDA* dst_vertex,
                                                 VertexId_CUDA vertex_size, VertexId_CUDA label_size) {
#if CUDA_ENABLE
  global_copy_mulilabel_move_gpu_kernel<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(
      dev_label, global_dev_label, dst_vertex, vertex_size, label_size);
  this->CUDA_DEVICE_SYNCHRONIZE();
#else
  printf("CUDA DISABLED Cuda_Stream::zero_copy_feature_move_gpu\n");
  exit(0);
#endif
}

void Cuda_Stream::global_copy_label_move_gpu(long* dev_label, long* global_dev_label, VertexId_CUDA* dst_vertex,
                                             VertexId_CUDA vertex_size) {
#if CUDA_ENABLE
  global_copy_label_move_gpu_kernel<<<CUDA_NUM_BLOCKS, CUDA_NUM_THREADS, 0, stream>>>(dev_label, global_dev_label,
                                                                                      dst_vertex, vertex_size);
  this->CUDA_DEVICE_SYNCHRONIZE();
#else
  printf("CUDA DISABLED Cuda_Stream::global_copy_label_move_gpu\n");
  exit(0);
#endif
}

void Cuda_Stream::Edge_Softmax_Forward_Block(float* msg_output, float* msg_input,  // data
                                             float* msg_cached, VertexId_CUDA* row_indices,
                                             VertexId_CUDA* column_offset, VertexId_CUDA batch_size,
                                             VertexId_CUDA feature_size) {
#if CUDA_ENABLE
  edge_softmax_forward_block<float, VertexId_CUDA><<<CUDA_NUM_BLOCKS_SOFTMAX, CUDA_NUM_THREADS_SOFTMAX, 0, stream>>>(
      msg_output, msg_input, msg_cached, row_indices, column_offset, batch_size, feature_size);
#else
  printf("CUDA DISABLED Cuda_Stream::Edge_Softmax_Forward_Block\n");
  exit(0);
#endif
}

void Cuda_Stream::Edge_Softmax_Backward_Block(float* msg_input_grad, float* msg_output_grad,  // data
                                              float* msg_cached, VertexId_CUDA* row_indices,
                                              VertexId_CUDA* column_offset, VertexId_CUDA batch_size,
                                              VertexId_CUDA feature_size) {
#if CUDA_ENABLE
  // printf("CUDA_DEBUGE_INFO:FORWARD RUN_SYNC with \t BLOCK_SIZE:%d\tfeature_size:%d\n",BLOCK_SIZE,feature_size);
  edge_softmax_backward_block<float, VertexId_CUDA><<<CUDA_NUM_BLOCKS_SOFTMAX, CUDA_NUM_THREADS_SOFTMAX, 0, stream>>>(
      msg_input_grad, msg_output_grad, msg_cached, row_indices, column_offset, batch_size, feature_size);
#else
  printf("CUDA DISABLED Cuda_Stream::Edge_Softmax_Backward_Block\n");
  exit(0);
#endif
}

void move_result_out(float* output, float* input, int src, int dst, int feature_size, bool sync) {
#if CUDA_ENABLE
  CHECK_CUDA_RESULT(
      hipMemcpy(output, input, ((long)(dst - src)) * feature_size * (sizeof(int)), hipMemcpyDeviceToHost));
  if (sync) hipDeviceSynchronize();
#else
  printf("CUDA DISABLED Cuda_Stream::Gather_By_Dst_From_Message\n");
  exit(0);
#endif
}

void move_data_in(float* d_pointer, float* h_pointer, int start, int end, int feature_size, bool sync) {
#if CUDA_ENABLE
  CHECK_CUDA_RESULT(
      hipMemcpy(d_pointer, h_pointer, ((long)(end - start)) * feature_size * (sizeof(float)), hipMemcpyHostToDevice));
  if (sync) hipDeviceSynchronize();
#else
  printf("CUDA DISABLED Cuda_Stream::Gather_By_Dst_From_Message\n");
  exit(0);
#endif
}

void move_edge_in(VertexId_CUDA* d_pointer, VertexId_CUDA* h_pointer, VertexId_CUDA start, VertexId_CUDA end,
                  int feature_size, bool sync) {
#if CUDA_ENABLE
  CHECK_CUDA_RESULT(hipMemcpy(d_pointer, h_pointer, ((long)(end - start)) * feature_size * (sizeof(VertexId_CUDA)),
                               hipMemcpyHostToDevice));
  if (sync) hipDeviceSynchronize();
#else
  printf("CUDA DISABLED move_edge_in\n");
  exit(0);
#endif
}
void move_bytes_in(void* d_pointer, void* h_pointer, long bytes, bool sync) {
#if CUDA_ENABLE
  CHECK_CUDA_RESULT(hipMemcpy(d_pointer, h_pointer, bytes, hipMemcpyHostToDevice));
  if (sync) hipDeviceSynchronize();
#else
  printf("CUDA DISABLED move_bytes_in\n");
  exit(0);
#endif
}

// void aggregate_comm_result(float* aggregate_buffer,float *input_buffer,int data_size,int feature_size,int
// partition_offset,bool sync){ #if CUDA_ENABLE
//    const int THREAD_SIZE=512;//getThreadNum(_meta->get_feature_size());
//    const int BLOCK_SIZE=32;
//    aggregate_data_buffer<<<THREAD_SIZE,BLOCK_SIZE>>>(aggregate_buffer,input_buffer,data_size,feature_size,partition_offset,sync);
//    if(sync)
//    	hipDeviceSynchronize();
//#else
//       printf("CUDA DISABLED aggregate_comm_result\n");
//       exit(0);
//#endif
//
//}

void ntsFreeHost(void* buffer) {
#if CUDA_ENABLE
  hipHostFree(buffer);
#else
  printf("CUDA DISABLED FreeBuffer\n");
  exit(0);
#endif
}

void FreeBuffer(float* buffer) {
#if CUDA_ENABLE
  hipFree(buffer);
#else
  printf("CUDA DISABLED FreeBuffer\n");
  exit(0);
#endif
}

void FreeEdge(VertexId_CUDA* buffer) {
#if CUDA_ENABLE
  hipFree(buffer);
#else
  printf("CUDA DISABLED FreeEdge\n");
  exit(0);
#endif
}
void zero_buffer(float* buffer, int size) {
#if CUDA_ENABLE
  CHECK_CUDA_RESULT(hipMemset(buffer, 0, sizeof(float) * size));
  hipDeviceSynchronize();
#else
  printf("CUDA DISABLED zero_buffer\n");
  exit(0);
#endif
}

template <typename T>
void free_gpu_data(T* data) {
#if CUDA_ENABLE
  hipFree(data);
#else
  printf("CUDA DISABLED free_gpu_data\n");
  exit(0);
#endif
}

template <typename T>
void alloc_gpu_data(T** input, int size) {
#if CUDA_ENABLE
  CHECK_CUDA_RESULT(hipMalloc(input, sizeof(T) * (size)));
#else
  printf("CUDA DISABLED Cuda_Stream::Gather_By_Dst_From_Message\n");
  exit(0);
#endif
}

void allocate_gpu_buffer(float** input, int size) {
#if CUDA_ENABLE
  CHECK_CUDA_RESULT(hipMalloc(input, sizeof(float) * (size)));
#else
  printf("CUDA DISABLED Cuda_Stream::Gather_By_Dst_From_Message\n");
  exit(0);
#endif
}

void allocate_gpu_edge(VertexId_CUDA** input, int size) {
#if CUDA_ENABLE
  CHECK_CUDA_RESULT(hipMalloc(input, sizeof(VertexId_CUDA) * (size)));
#else
  printf("CUDA DISABLED Cuda_Stream::Gather_By_Dst_From_Message\n");
  exit(0);

#endif
}
